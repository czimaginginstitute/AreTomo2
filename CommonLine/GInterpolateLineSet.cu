#include "hip/hip_runtime.h"
#include "CCommonLineInc.h"
#include "../Util/CUtilInc.h"
#include <memory.h>
#include <stdio.h>
#include <math.h>

using namespace CommonLine;

static __global__ void mGInterpolate
(	hipfftComplex* gCmpLine1,
	hipfftComplex* gCmpLine2,
	int iCmpSize,
	float fWeight,
	hipfftComplex* gCmpResult
)
{	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= iCmpSize) return;
	gCmpResult[i].x = fWeight * gCmpLine1[i].x
		+ (1 - fWeight) * gCmpLine2[i].x;
	gCmpResult[i].y = fWeight * gCmpLine1[i].y
		+ (1 - fWeight) * gCmpLine2[i].y;
}

static CPossibleLines* s_pPossibleLines = 0L;
static float* s_pfRotAngles = 0L;
static CLineSet* s_pLineSet = 0L;

void GInterpolateLineSet::DoIt
(	CPossibleLines* pPossibleLines,
	float* pfRotAngles,
	CLineSet* pLineSet
)
{	s_pPossibleLines = pPossibleLines;
	s_pfRotAngles = pfRotAngles;
	s_pLineSet = pLineSet;
	//--------------------
	int iNumGpus = s_pLineSet->m_iNumGpus;
	GInterpolateLineSet* pThreads = new GInterpolateLineSet[iNumGpus];
	for(int i=0; i<iNumGpus; i++)
	{	pThreads[i].Run(i);
	}
	//-------------------------
	for(int i=0; i<iNumGpus; i++)
	{	pThreads[i].WaitForExit(-1.0f);
	}
	delete[] pThreads;
}

GInterpolateLineSet::GInterpolateLineSet(void)
{
	m_gCmpLine1 = 0L;
	m_gCmpLine2 = 0L;
}

GInterpolateLineSet::~GInterpolateLineSet(void)
{
	this->Clean();
}

void GInterpolateLineSet::Clean(void)
{
	if(m_gCmpLine1 != 0L) hipFree(m_gCmpLine1);
	if(m_gCmpLine2 != 0L) hipFree(m_gCmpLine2);
	m_gCmpLine1 = 0L;
	m_gCmpLine2 = 0L;
}

void GInterpolateLineSet::Run(int iThreadID)
{	
	m_iGpuID = s_pLineSet->GetGpuID(iThreadID);
	m_iCmpSize = s_pLineSet->m_iCmpSize;
	this->Start();
}

void GInterpolateLineSet::ThreadMain(void)
{
	hipSetDevice(m_iGpuID);
	//----------------------
	size_t tBytes = sizeof(hipfftComplex) * m_iCmpSize;
	hipMalloc(&m_gCmpLine1, tBytes);
	hipMalloc(&m_gCmpLine2, tBytes);
	//-------------------------------
	GFunctions aGFunctions;
	for(int i=0; i<s_pLineSet->m_iNumProjs; i++)
	{	int iGpuID = s_pLineSet->GetLineGpu(i);
		if(iGpuID != m_iGpuID) continue;
		mInterpolate(i);
	}
}

void GInterpolateLineSet::mInterpolate(int iProj)
{
	int iNumLines = s_pPossibleLines->m_iNumLines;
	float fRotAngle = s_pfRotAngles[iProj];
	//-------------------------------------
	float fLine = s_pPossibleLines->CalcLinePos(fRotAngle);
	int iLine1 = (int)fLine;
	if(iLine1 < 0) iLine1 = 0;
	//------------------------
	int iLine2 = iLine1 + 1;
	if(iLine2 >= iNumLines) 
	{	iLine2 = iNumLines - 1;
		iLine1 = iLine2 - 1;
	}
	//--------------------------
	float fW = 1.0f - (fLine - iLine1);
	if(iLine1 == 0) fW = 1.0f;
	else if(iLine2 == (iNumLines - 1)) fW = 0.0f;
	//-------------------------------------------
	s_pPossibleLines->GetLine(iProj, iLine1, m_gCmpLine1);
	s_pPossibleLines->GetLine(iProj, iLine2, m_gCmpLine2);
	//----------------------------------------------------
	hipfftComplex* gCmpRes = s_pLineSet->GetLine(iProj);
	dim3 aBlockDim(512, 1);
	dim3 aGridDim(m_iCmpSize / aBlockDim.x + 1, 1);
	mGInterpolate<<<aGridDim, aBlockDim>>>
	( m_gCmpLine1, m_gCmpLine2, m_iCmpSize,
	  fW, gCmpRes
	);
}
