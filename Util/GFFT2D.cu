#include "hip/hip_runtime.h"
#include "CUtilInc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <memory.h>
#include <math.h>
#include <stdio.h>

using namespace Util;

static __global__ void mGMultiply
(	hipfftComplex* gCmp, int iCmpY,
	float fFactor
)
{	int y = blockIdx.y * blockDim.y + threadIdx.y;
        if(y >= iCmpY) return;
        int i = y * gridDim.x + blockIdx.x;
	gCmp[i].x *= fFactor;
	gCmp[i].y *= fFactor;
}

static __global__ void mGRemoveAmp(hipfftComplex* gCmp, int iCmpY)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(y >= iCmpY) return;
	int i = y * gridDim.x + blockIdx.x;
	//---------------------------------
	float fRe = gCmp[i].x;
	float fIm = gCmp[i].y;
	fRe = sqrtf(fRe * fRe + fIm * fIm) + (float)1e-20;
	gCmp[i].x /= fRe;
	gCmp[i].y /= fRe;
}

GFFT2D::GFFT2D(void)
{
	m_aStream = (hipStream_t)0;
}

GFFT2D::~GFFT2D(void)
{
}

void GFFT2D::SetStream(hipStream_t stream)
{
	m_aStream = stream;
}

void GFFT2D::DestroyPlan(void)
{
	if(m_cufftPlan == 0) return;
	hipfftDestroy(m_cufftPlan);
	m_cufftPlan = 0;
	m_aiFFTSize[0] = 0;
	m_aiFFTSize[1] = 0;
	m_cufftType = HIPFFT_R2C;
}

void GFFT2D::CreatePlan(int* piFFTSize, bool bForward)
{	
	hipfftType fftType = bForward ? HIPFFT_R2C : HIPFFT_C2R;
	if(m_cufftType != fftType) this->DestroyPlan();
	else if(m_aiFFTSize[0] != piFFTSize[0]) this->DestroyPlan();
	else if(m_aiFFTSize[1] != piFFTSize[1]) this->DestroyPlan();
	if(m_cufftPlan != 0) return;
	//--------------------------
	m_cufftType = fftType;
	m_aiFFTSize[0] = piFFTSize[0];
	m_aiFFTSize[1] = piFFTSize[1];
	hipfftResult res = hipfftPlan2d(&m_cufftPlan, 
	   m_aiFFTSize[1], m_aiFFTSize[0], m_cufftType);
	mCheckError(res, "GFFT2D::CreatePlan");
}

void GFFT2D::Forward(float* gfPadImg, bool bNorm)
{
	hipfftSetStream(m_cufftPlan, m_aStream);	
	hipfftResult res = hipfftExecR2C(m_cufftPlan, 
	   (hipfftReal*)gfPadImg, (hipfftComplex*)gfPadImg);
	if(bNorm) mNormalize((hipfftComplex*)gfPadImg);
	mCheckError(res, "GFFT2D::Forward 1");
}

void GFFT2D::Forward(float* gfImg, hipfftComplex* gCmp, bool bNorm)
{
	hipfftSetStream(m_cufftPlan, m_aStream);
	hipfftResult res = hipfftExecR2C(m_cufftPlan,
	   (hipfftReal*)gfImg, gCmp);
	if(bNorm) mNormalize(gCmp);
	mCheckError(res, "GFFT2D::Forward 2");
}

void GFFT2D::Inverse(hipfftComplex* gCmp)
{
	hipfftSetStream(m_cufftPlan, m_aStream);	
	hipfftResult res = hipfftExecC2R(m_cufftPlan, gCmp, 
	   (hipfftReal*)gCmp);
	mCheckError(res, "GFFT2D::Inverse 1");
}

void GFFT2D::Inverse(hipfftComplex* gCmpImg, float* gfImg)
{
	hipfftSetStream(m_cufftPlan, m_aStream);
	hipfftResult res = hipfftExecC2R(m_cufftPlan, gCmpImg,
	   (hipfftReal*)gfImg);
	mCheckError(res, "GFFT2D::Inverse 2");
}

void GFFT2D::RemoveAmp(hipfftComplex* gCmp, int* piCmpSize)
{
	dim3 aBlockDim(1, 512);
	dim3 aGridDim(piCmpSize[0], piCmpSize[1] / aBlockDim.y + 1);
	mGRemoveAmp<<<aGridDim, aBlockDim, 0, m_aStream>>>(gCmp, piCmpSize[1]);
}

void GFFT2D::mNormalize(hipfftComplex* gCmpImg)
{
	int iCmpSizeX = m_aiFFTSize[0] / 2 + 1;
	dim3 aBlockDim(1, 512);
	dim3 aGridDim(iCmpSizeX, 1);
	aGridDim.y = (m_aiFFTSize[1] + aBlockDim.y - 1) / aBlockDim.y;
	float fFactor = (1.0f / m_aiFFTSize[0]) / m_aiFFTSize[1];
	mGMultiply<<<aGridDim, aBlockDim, 0, m_aStream>>>(gCmpImg, 
	   m_aiFFTSize[1], fFactor);
}

void GFFT2D::mCheckError(hipfftResult error, const char* pcFunc)
{
	switch(error)
	{	case HIPFFT_SUCCESS: 
		return;
		//-----------------
		case HIPFFT_INVALID_PLAN:
		printf("%s: HIPFFT_INVALID_PLAN\n\n", pcFunc);
		//-------------------------------------------
		case HIPFFT_ALLOC_FAILED: 
		printf("%s: HIPFFT_ALLOC_FAILED\n\n", pcFunc);
		//-------------------------------------------
		case HIPFFT_INVALID_TYPE:
                printf("%s: HIPFFT_INVALID_TYPE\n\n", pcFunc);
                //-------------------------------------------
		case HIPFFT_INVALID_VALUE:
		printf("%s: HIPFFT_INVALID_VALUE\n\n", pcFunc);
                //--------------------------------------------
		case HIPFFT_INTERNAL_ERROR:
		printf("%s: HIPFFT_INTERNAL_ERROR\n\n", pcFunc);
		//---------------------------------------------
		case HIPFFT_EXEC_FAILED:
		printf("%s: HIPFFT_EXEC_FAILED\n\n", pcFunc);
		//------------------------------------------
		case HIPFFT_SETUP_FAILED:
		printf("%s: HIPFFT_SETUP_FAILED\n\n", pcFunc);
		//------------------------------------------
		case HIPFFT_INVALID_SIZE:
		printf("%s: HIPFFT_INVALID_SIZE\n\n", pcFunc);
		//-------------------------------------------
		case HIPFFT_UNALIGNED_DATA:
		printf("%s: HIPFFT_UNALIGNED_DATA\n\n", pcFunc);
        }
}
